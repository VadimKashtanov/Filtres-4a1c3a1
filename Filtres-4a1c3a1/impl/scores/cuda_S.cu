#include "hip/hip_runtime.h"
#include "S.cuh"

#include "../../impl_tmpl/tmpl_etc.cu"

#define pseudo_alea_d_une_grain(i) ((float)((121+(i%1234))*31 % 1001 ) / 1001.0)

//	===============================================================

static __global__ void kerd_nvidia_score_somme(
	float * y, uint depart, uint T,
	float * score, float * _PRIXS)
{
	uint t = threadIdx.x + blockIdx.x + blockDim.x;
	if (t < T) {
		float s = 0;
		FOR(0, p, P) {
			float _y = y[(0 + t)*P + p];
			float alea = 2*pseudo_alea_d_une_grain(t + ((uint)_y % 10001))-1;
			_y += alea * SCORE_Y_COEF_BRUIT;
			s += (P-p)*cuda_SCORE(
				_y, _PRIXS[depart+t+p+1], _PRIXS[depart+t], alea * SCORE_Y_COEF_BRUIT
			);
		}
		atomicAdd(score, s);
	}
};

float nvidia_somme_score(float * y, uint depart, uint T)
{
	float * somme_score__d = cudalloc<float>(1);
	CONTROLE_CUDA(hipMemset(somme_score__d, 0, sizeof(float)*1));
	kerd_nvidia_score_somme<<<dim3(KERD(T,1)),dim3(1)>>>(
		y, depart, T,
		somme_score__d, prixs__d
	);
	ATTENDRE_CUDA();
	float somme_score;
	CONTROLE_CUDA(hipMemcpy(&somme_score, somme_score__d, sizeof(float)*1, hipMemcpyDeviceToHost));
	CONTROLE_CUDA(hipFree(somme_score__d));
	return somme_score;
};

float  nvidia_score_finale(float somme, uint T) {
	return APRES_SCORE(somme / (float)(P * T));
};

//	===============================================================

float d_nvidia_score_finale(float somme, uint T) {
	return dAPRES_SCORE(somme / (float)(P * T)) / (float)(P * T);
};

//	===============================================================

static __global__ void kerd_nvidia_score_dpowf(
	float _dy, float * y, float * dy,
	uint depart, uint T,
	float * _PRIXS)
{
	uint _t = threadIdx.x + blockIdx.x * blockDim.x;

	if (_t < T) {
		FOR(0, p, P) {
			float _y = y[(0+_t)*P+p];
			float alea = 2*pseudo_alea_d_une_grain(_t + ((uint)_y % 10001))-1;
			dy[(0+_t)*P+p] = _dy * (P-p)*cuda_dSCORE(
				y[(0+_t)*P+p]+alea*SCORE_Y_COEF_BRUIT, _PRIXS[depart+_t+p+1], _PRIXS[depart+_t/*+p*/], alea*SCORE_Y_COEF_BRUIT
			);
		}
	}
};

void d_nvidia_somme_score(float d_score, float * y, float * dy, uint depart, uint T) {
	kerd_nvidia_score_dpowf<<<dim3(KERD(T,1024)), dim3(1024)>>>(
		d_score,
		y, dy,
		depart, T,
		prixs__d
	);
	ATTENDRE_CUDA();
};