#include "hip/hip_runtime.h"
#include "mdl.cuh"

#include "../../impl_tmpl/tmpl_etc.cu"

float mdl_les_gains(Mdl_t * mdl, uint t0, uint t1, uint mode, float GRAND_COEF) {
	ASSERT(GRAND_COEF >= 2);
	ASSERT(mdl->T == (t1-t0));
	float * _y = gpu_vers_cpu<float>(mdl->y__d[C-1], (t1-t0)*P);
	float somme = 0;
	float potentiel = 0;
	FOR(t0, t, t1) {
		somme     += powf(fabs(prixs[t+1]/prixs[t]-1),GRAND_COEF) * (signe((prixs[t+1]/prixs[t]-1)) == signe(_y[(t-t0)*P+0]));
		potentiel += powf(fabs(prixs[t+1]/prixs[t]-1),GRAND_COEF);
	}
	free(_y);
	return somme / potentiel;
};

float mdl_score(Mdl_t * mdl, uint t0, uint t1, uint mode) {
	ASSERT(mdl->T == (t1-t0));
	if (mode == 0) mdl_zero_cpu(mdl);
	else           mdl_zero_gpu(mdl);
	//
	mdl_f(mdl, t0, t1, mode);
	//
	float somme_score;
	if (mode == 0) somme_score =  intel_somme_score(mdl->y[C-1],    t0, (t1-t0));
	else           somme_score = nvidia_somme_score(mdl->y__d[C-1], t0, (t1-t0));
	//
	if (mode == 0) return  intel_score_finale(somme_score, (t1-t0));
	else           return nvidia_score_finale(somme_score, (t1-t0));
};

float* mdl_pred(Mdl_t * mdl, uint t0, uint t1, uint mode) {
	ASSERT(mdl->T == (t1-t0));
	if (mode == 0) mdl_zero_cpu(mdl);
	else           mdl_zero_gpu(mdl);
	//
	mdl_f(mdl, t0, t1, mode);
	if (mode == 0) return  intel_prediction(mdl->y[C-1], t0, (t1-t0));
	else           return nvidia_prediction(mdl->y__d[C-1], t0, (t1-t0));
};

void mdl_aller_retour(Mdl_t * mdl, uint t0, uint t1, uint mode) {
	ASSERT(mdl->T == (t1-t0));
	if (mode == 0) mdl_zero_cpu(mdl);
	else           mdl_zero_gpu(mdl);
	mdl_f(mdl, t0, t1, mode);
	//
	float somme_score;
	if (mode == 0) somme_score =  intel_somme_score(mdl->y[C-1], t0, (t1-t0));
	else           somme_score = nvidia_somme_score(mdl->y__d[C-1], t0, (t1-t0));
	//
	float d_score;
	if (mode == 0) d_score =  d_intel_score_finale(somme_score, (t1-t0));
	else           d_score = d_nvidia_score_finale(somme_score, (t1-t0));
	//
	if (mode == 0)  d_intel_somme_score(d_score, mdl->y[C-1],    mdl->dy[C-1], t0, (t1-t0));
	else           d_nvidia_somme_score(d_score, mdl->y__d[C-1], mdl->dy__d[C-1], t0, (t1-t0));
	mdl_df(mdl, t0, t1, mode);
};