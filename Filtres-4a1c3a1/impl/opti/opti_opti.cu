#include "hip/hip_runtime.h"
#include "opti.cuh"

#include "../../impl_tmpl/tmpl_etc.cu"

static uint * tout_zeroiser = UNIFORME_C(1);

PAS_OPTIMISER()
void __interne_optimiser(
	Mdl_t * mdl,
	uint t0, uint t1,
	float * alpha, float div,
	uint methode, uint I,
	uint ** masque,
	uint PERTURBATIONS,
	uint zero_accumulation_tous_les[C])
{
	mdl_zero_deriv_gpu(mdl, tout_zeroiser);
	//
	//	Cree les listes pour les `hist` si un opti en a besoin 
	Opti_classe_t opti_classe;
	if      (methode == SGD)     opti_classe.sgd     = (uint)NULL;
	else if (methode == RMSPROP) opti_classe.rmsprop = cree_rmsprop(mdl);
	else if (methode == ADAM)    opti_classe.adam    = cree_adam(mdl);
	else ERR("Pas de methode %i d'optimisation", methode);
	
	//	Plumer grad pour mieux y voire
	mdl_plume_grad(mdl, t0, t1);
	
	/* ------- Optimisation ----------- */
	uint zeroiser[C];
	FOR(0, i, I) {
		//
		FOR(0, j, C) {
			if (i % zero_accumulation_tous_les[j] == 0)
				zeroiser[j] = 1;
			else
				zeroiser[j] = 0;
		}
		//
		perturber(mdl, PERTURBATIONS);
		mdl_aller_retour(mdl, t0, t1, 3);
		
		//	--------- * Optimisation * -------------
#define optimiser_la_couche zeroiser
		if (methode == SGD)     opti_simple (zero_accumulation_tous_les, optimiser_la_couche, mdl, alpha, div, masque);
		if (methode == RMSPROP) opti_rmsprop(zero_accumulation_tous_les, optimiser_la_couche, mdl, opti_classe.rmsprop, alpha, div, masque);
		if (methode == ADAM)    opti_adam   (zero_accumulation_tous_les, optimiser_la_couche, mdl, opti_classe.adam,    alpha, div, masque);
		//
		mdl_zero_deriv_gpu(mdl, zeroiser);
		//
		mdl_normer_les_filtres(mdl);
		//
		if (i % /*5*/100 == 0) {
			float* __pred = mdl_pred(mdl, t0, t1, 3);
			float  _score = mdl_score(mdl, t0, t1, 3);
			//
			float les_gains__2 = mdl_les_gains(mdl, t0, t1, 3,   2.0);
			float les_gains__4 = mdl_les_gains(mdl, t0, t1, 3,   4.0);
			//
			printf("%3.i/%3.i| perf={", i, I);
			FOR(0, p, P) printf("%+f%%, ", 100*__pred[p]);
			free(__pred);
			printf("} score=\033[93m%+f\033[0m (%%.potentiel^2=%+f, %%.potentiel^4=%+f)\n",
				_score,
				les_gains__2,
				les_gains__4
			);
			if (fabs(_score) < 0.00001) {
				printf("Score < 0.00001 => Fin d'optimisation\n");
				break;
			}
		}
	}

	//	Liberer
	if      (methode == SGD)     opti_classe.sgd = 0;
	else if (methode == RMSPROP) liberer_rmsprop(opti_classe.rmsprop);
	else if (methode == ADAM)    liberer_adam   (opti_classe.adam   );
};

void optimiser(
	Mdl_t * mdl,
	uint t0, uint t1,
	float * alpha, float div,
	uint methode, uint I,
	float * pourcent_masque,
	uint PERTURBATIONS,
	uint zero_accumulation_tous_les[C])
{
	Masque_t * masque = cree_masque(mdl, pourcent_masque);
	//
	__interne_optimiser(
		mdl,
		t0, t1,
		alpha, div,
		methode, I,
		masque->masque,
		PERTURBATIONS,
		zero_accumulation_tous_les);
	//
	sortire_masque(mdl, masque);
};